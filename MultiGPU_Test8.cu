#include "hip/hip_runtime.h"
#include "Utilities.cuh"
#include "InputOutput.cuh"

#define BLOCKSIZE 128

/*******************/
/* KERNEL FUNCTION */
/*******************/
template<class T>
__global__ void kernelFunction(T * __restrict__ d_data, const unsigned int NperGPU) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < NperGPU) for (int k = 0; k < 1000; k++) d_data[tid] = d_data[tid] * d_data[tid];

}

/******************/
/* PLAN STRUCTURE */
/******************/
// --- Async
template<class T>
struct plan {
	T 				*d_data;
};

/*********************/
/* SVD PLAN CREATION */
/*********************/
template<class T>
void createPlan(plan<T>& plan, unsigned int NperGPU, unsigned int gpuID) {

    // --- Device allocation
    gpuErrchk(hipSetDevice(gpuID));
	gpuErrchk(hipMalloc(&(plan.d_data), NperGPU * sizeof(T)));
}

/********/
/* MAIN */
/********/
int main() {

    const int numGPUs 	= 4;
    const int NperGPU 	= 500000;
    const int N			= NperGPU * numGPUs;

    plan<double> plan[numGPUs];
    for (int k = 0; k < numGPUs; k++) createPlan(plan[k], NperGPU, k);

    // --- "Breadth-first" approach - async
    double *inputMatrices;   gpuErrchk(hipHostMalloc(&inputMatrices, N * sizeof(double)));
	for (int k = 0; k < numGPUs; k++) {
 		gpuErrchk(hipSetDevice(k));
 		gpuErrchk(hipMemcpyAsync(plan[k].d_data, inputMatrices + k * NperGPU, NperGPU * sizeof(double), hipMemcpyHostToDevice));
 	}

 	for (int k = 0; k < numGPUs; k++) {
 		gpuErrchk(hipSetDevice(k));
 		kernelFunction<<<iDivUp(NperGPU, BLOCKSIZE), BLOCKSIZE>>>(plan[k].d_data, NperGPU);
 	}

 	for (int k = 0; k < numGPUs; k++) {
 		gpuErrchk(hipSetDevice(k));
 		gpuErrchk(hipMemcpyAsync(inputMatrices + k * NperGPU, plan[k].d_data, NperGPU * sizeof(double), hipMemcpyDeviceToHost));
 	}

    gpuErrchk(hipDeviceReset());
}

