#include "hip/hip_runtime.h"
#include "Utilities.cuh"
#include "InputOutput.cuh"

#define BLOCKSIZE 128

/*******************/
/* KERNEL FUNCTION */
/*******************/
template<class T>
__global__ void kernelFunction(T * __restrict__ d_data, const unsigned int NperGPU) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < NperGPU) for (int k = 0; k < 1000; k++) d_data[tid] = d_data[tid] * d_data[tid];

}

/******************/
/* PLAN STRUCTURE */
/******************/
template<class T>
struct plan {
	T 				*d_data;
	T 				*h_data;
};

/*********************/
/* SVD PLAN CREATION */
/*********************/
template<class T>
void createPlan(plan<T>& plan, unsigned int NperGPU, unsigned int gpuID) {

    // --- Device allocation
    gpuErrchk(hipSetDevice(gpuID));
	gpuErrchk(hipMalloc(&(plan.d_data), NperGPU * sizeof(T)));
	gpuErrchk(hipHostMalloc((void **)&plan.h_data, NperGPU * sizeof(T)));
}

/********/
/* MAIN */
/********/
int main() {

    const int numGPUs 	= 4;
    const int NperGPU 	= 500000;
    const int N			= NperGPU * numGPUs;

    plan<double> plan[numGPUs];
    for (int k = 0; k < numGPUs; k++) createPlan(plan[k], NperGPU, k);

    // --- "Depth-first" approach - no stream
    for (int k = 0; k < numGPUs; k++)
    {
      	gpuErrchk(hipSetDevice(k));
        gpuErrchk(hipMemcpyAsync(plan[k].d_data, plan[k].h_data, NperGPU * sizeof(double), hipMemcpyHostToDevice));
        kernelFunction<<<iDivUp(NperGPU, BLOCKSIZE), BLOCKSIZE>>>(plan[k].d_data, NperGPU);
        gpuErrchk(hipMemcpyAsync(plan[k].h_data, plan[k].d_data, NperGPU * sizeof(double), hipMemcpyDeviceToHost));
    }

    gpuErrchk(hipDeviceReset());
}

